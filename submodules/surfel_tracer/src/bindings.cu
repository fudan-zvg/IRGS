#include <iostream>
#include <string>
#include <vector>

#include <cstdint>
#include <cmath>
#include <cassert>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <glm/glm.hpp>

#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#include <gtracer/bvh.h>

namespace py = pybind11;
namespace gtracer {

class GaussianTracer {
public:
    GaussianTracer(){
        triangle_bvh = TriangleBvhBase::make();
    }

    void build_bvh(const torch::Tensor& triangles){
        const size_t n_triangles = triangles.size(0);
        hipStream_t m_stream = at::cuda::getCurrentCUDAStream();;
        triangle_bvh->build_bvh(triangles.data_ptr<float>(), n_triangles, m_stream);
    }

    void update_bvh(const torch::Tensor& triangles){
        const size_t n_triangles = triangles.size(0);
        hipStream_t m_stream = at::cuda::getCurrentCUDAStream();;
        triangle_bvh->update_bvh(triangles.data_ptr<float>(), n_triangles, m_stream);
    }

    void trace_forward(
        const torch::Tensor rays_o, const torch::Tensor rays_d, const torch::Tensor gs_idxs, 
        const torch::Tensor means3D, const torch::Tensor opacity, const torch::Tensor ru, const torch::Tensor rv, const torch::Tensor normals, const torch::Tensor features, const torch::Tensor shs, 
        torch::Tensor color, torch::Tensor normal, torch::Tensor feature, torch::Tensor depth, torch::Tensor alpha, 
        const float alpha_min, const float transmittance_min, const int deg, const bool back_culling
        ){
        const uint32_t n_elements = rays_o.size(0);
        hipStream_t stream = at::cuda::getCurrentCUDAStream();

        int max_coeffs = shs.size(1);
        int S = features.size(1);

        triangle_bvh->gaussian_trace_forward(
            n_elements, S, (const glm::vec3*)rays_o.data_ptr<float>(), (const glm::vec3*)rays_d.data_ptr<float>(), gs_idxs.data_ptr<int>(), 
            (const glm::vec3*)means3D.data_ptr<float>(), opacity.data_ptr<float>(), (const glm::vec3*)ru.data_ptr<float>(), (const glm::vec3*)rv.data_ptr<float>(), (const glm::vec3*)normals.data_ptr<float>(), features.data_ptr<float>(), (const glm::vec3*)shs.data_ptr<float>(), 
            (glm::vec3*)color.data_ptr<float>(), (glm::vec3*)normal.data_ptr<float>(), feature.data_ptr<float>(), depth.data_ptr<float>(), alpha.data_ptr<float>(), 
            alpha_min, transmittance_min, deg, max_coeffs, back_culling, stream);
    }
    
    void intersection_test(
        const torch::Tensor rays_o, const torch::Tensor rays_d, const torch::Tensor gs_idxs, 
        const torch::Tensor means3D, const torch::Tensor opacity, const torch::Tensor ru, const torch::Tensor rv, const torch::Tensor normals, torch::Tensor intersection
        ){
        const uint32_t n_elements = rays_o.size(0);
        hipStream_t stream = at::cuda::getCurrentCUDAStream();

        triangle_bvh->intersection_test(
            n_elements, (const glm::vec3*)rays_o.data_ptr<float>(), (const glm::vec3*)rays_d.data_ptr<float>(), gs_idxs.data_ptr<int>(), 
            (const glm::vec3*)means3D.data_ptr<float>(), opacity.data_ptr<float>(), (const glm::vec3*)ru.data_ptr<float>(), (const glm::vec3*)rv.data_ptr<float>(), (const glm::vec3*)normals.data_ptr<float>(), intersection.data_ptr<bool>(), stream);
    }
    
    void trace_backward(
        const torch::Tensor rays_o, const torch::Tensor rays_d, const torch::Tensor gs_idxs, 
        const torch::Tensor means3D, const torch::Tensor opacity, const torch::Tensor ru, const torch::Tensor rv, const torch::Tensor normals, const torch::Tensor features, const torch::Tensor shs, 
        const torch::Tensor color, const torch::Tensor normal, const torch::Tensor feature, const torch::Tensor depth, const torch::Tensor alpha, 
        torch::Tensor grad_rays_o, torch::Tensor grad_rays_d, torch::Tensor grad_means3D, torch::Tensor grad_opacity, torch::Tensor grad_ru, torch::Tensor grad_rv, torch::Tensor grad_normals, torch::Tensor grad_features, torch::Tensor grad_shs, 
        const torch::Tensor grad_out_color, const torch::Tensor grad_out_normal, const torch::Tensor grad_out_feature, const torch::Tensor grad_out_depth, const torch::Tensor grad_out_alpha,
        const float alpha_min, const float transmittance_min, const int deg, const bool back_culling
        ){
        const uint32_t n_elements = rays_o.size(0);
        hipStream_t stream = at::cuda::getCurrentCUDAStream();

        int max_coeffs = shs.size(1);
        int S = features.size(1);

        triangle_bvh->gaussian_trace_backward(
            n_elements, S, (const glm::vec3*)rays_o.data_ptr<float>(), (const glm::vec3*)rays_d.data_ptr<float>(), gs_idxs.data_ptr<int>(), 
            (const glm::vec3*)means3D.data_ptr<float>(), opacity.data_ptr<float>(), (const glm::vec3*)ru.data_ptr<float>(), (const glm::vec3*)rv.data_ptr<float>(), (const glm::vec3*)normals.data_ptr<float>(), features.data_ptr<float>(), (const glm::vec3*)shs.data_ptr<float>(), 
            (const glm::vec3*)color.data_ptr<float>(), (const glm::vec3*)normal.data_ptr<float>(), feature.data_ptr<float>(), depth.data_ptr<float>(), alpha.data_ptr<float>(), 
            (glm::vec3*)grad_rays_o.data_ptr<float>(), (glm::vec3*)grad_rays_d.data_ptr<float>(), (glm::vec3*)grad_means3D.data_ptr<float>(), grad_opacity.data_ptr<float>(), (glm::vec3*)grad_ru.data_ptr<float>(), (glm::vec3*)grad_rv.data_ptr<float>(), (glm::vec3*)grad_normals.data_ptr<float>(), grad_features.data_ptr<float>(), (glm::vec3*)grad_shs.data_ptr<float>(), 
            (const glm::vec3*)grad_out_color.data_ptr<float>(), (const glm::vec3*)grad_out_normal.data_ptr<float>(), grad_out_feature.data_ptr<float>(), grad_out_depth.data_ptr<float>(), grad_out_alpha.data_ptr<float>(),
            alpha_min, transmittance_min, deg, max_coeffs, back_culling, stream);
    }

    std::shared_ptr<TriangleBvhBase> triangle_bvh;
};

GaussianTracer* create_gaussiantracer() {
    return new GaussianTracer{};
}

}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {

py::class_<gtracer::GaussianTracer>(m, "GaussianTracer")
    .def("intersection_test", &gtracer::GaussianTracer::intersection_test)
    .def("trace_forward", &gtracer::GaussianTracer::trace_forward)
    .def("trace_backward", &gtracer::GaussianTracer::trace_backward)
    .def("build_bvh", &gtracer::GaussianTracer::build_bvh)
    .def("update_bvh", &gtracer::GaussianTracer::update_bvh);

m.def("create_gaussiantracer", &gtracer::create_gaussiantracer);

}